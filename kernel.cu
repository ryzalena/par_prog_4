#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <iostream>

__global__ void multiplyMatricesKernel(int* matrix1, int* matrix2, int* result_matrix, int size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < size && col < size) {
        int value = 0;
        for (int k = 0; k < size; ++k) {
            value += matrix1[row * size + k] * matrix2[k * size + col];
        }
        result_matrix[row * size + col] = value;
    }
}

void generateRandomMatrix(int* matrix, int size) {
    for (int i = 0; i < size * size; ++i) {
        matrix[i] = rand() % 100;
    }
}

void multiplyMatrices(int* matrix1, int* matrix2, int* result_matrix, int size) {
    int* d_matrix1;
    int* d_matrix2;
    int* d_result_matrix;
    int rank = 128;

    hipMalloc(&d_matrix1, size * size * sizeof(int));
    hipMalloc(&d_matrix2, size * size * sizeof(int));
    hipMalloc(&d_result_matrix, size * size * sizeof(int));

    hipMemcpy(d_matrix1, matrix1, size * size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_matrix2, matrix2, size * size * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(rank, rank);
    dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    multiplyMatricesKernel << <numBlocks, threadsPerBlock >> > (d_matrix1, d_matrix2, d_result_matrix, size);

    hipMemcpy(result_matrix, d_result_matrix, size * size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_matrix1);
    hipFree(d_matrix2);
    hipFree(d_result_matrix);
}

void writeMatrixToFile(const char* filename, int* matrix, int size) {
    FILE* file = fopen(filename, "w");
    if (file == NULL) {
        printf("Не удалось открыть файл: %s\n", filename);
        exit(1);
    }

    fprintf(file, "%d\n", size);
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            fprintf(file, "%d ", matrix[i * size + j]);
        }
        fprintf(file, "\n");
    }

    fclose(file);
}

void writeTimeToFile(double computation_time, const char* filename) {
    FILE* file = fopen(filename, "a");
    if (file == NULL) {
        printf("Не удалось открыть файл: %s\n", filename);
        exit(1);
    }

    fprintf(file, "%.5f\n", computation_time);
    fclose(file);
}

void writeTaskSizeToFile(int size, long long task_size, const char* filename) {
    FILE* file = fopen(filename, "a");
    if (file == NULL) {
        printf("Не удалось открыть файл: %s\n", filename);
        exit(1);
    }

    fprintf(file, "%d\n%lld\n", size, task_size);
    fclose(file);
}

int main() {
    setlocale(LC_ALL, "ru_RU.UTF-8");

    int N = 500;
    const char* file_stat = "result.txt";

    srand((unsigned int)time(NULL));

    while (N <= 3000) {
        long long task_size = (long long)N * N * N;
        writeTaskSizeToFile(N, task_size, file_stat);

        printf("Размер матриц %dx%d\n", N, N);

        for (int i = 0; i < 10; ++i) {
            int* matrix1 = (int*)malloc(N * N * sizeof(int));
            int* matrix2 = (int*)malloc(N * N * sizeof(int));
            int* result_matrix = (int*)malloc(N * N * sizeof(int));

            if (!matrix1 || !matrix2 || !result_matrix) {
                printf("Ошибка выделения памяти\n");
                exit(1);
            }

            generateRandomMatrix(matrix1, N);
            generateRandomMatrix(matrix2, N);

            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);

            hipEventRecord(start);
            multiplyMatrices(matrix1, matrix2, result_matrix, N);
            hipEventRecord(stop);

            hipEventSynchronize(stop);

            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);

            printf("Матрицы перемножены.\n");
            printf("Время умножения матриц: %.5f мс\n", milliseconds);

            // записываем время в файл с точностью до 5 знаков после запятой
            writeTimeToFile((double)milliseconds, file_stat);

            free(matrix1);
            free(matrix2);
            free(result_matrix);

            hipEventDestroy(start);
            hipEventDestroy(stop);
        }

        N += 500;
    }

    return 0;
}